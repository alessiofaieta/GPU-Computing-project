#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>

#include <hipcub/hipcub.hpp>

#include "include/hip/hip_runtime_api.h"
#include "utils.h"


struct CompareOp {
	__device__ __host__ bool operator()(const int &a, const int &b) const {
		return a < b;
	}
};


int main(int argc, char** argv) {
	if (argc != 2) {
		std::cerr << "[ERROR] Wrong number of arguments" << std::endl;
		std::cerr << "USAGE: " << argv[0] << " VECTOR_SIZE_EXP" << std::endl;
		exit(1);
	}

	init_random_generator();

	size_t vector_size_exponent = atoi(argv[1]);
	size_t vector_size = std::pow(2, vector_size_exponent);
	std::cout << "Vector size: 2 ^ " << vector_size_exponent << " = " << vector_size << std::endl;
	size_t vector_allocated_bytes = vector_size * sizeof(int);
	const int n_allocated_vectors = 2;
	size_t total_allocated_bytes = vector_allocated_bytes * n_allocated_vectors;
	std::cout << "Allocating " << n_allocated_vectors << " * " << vector_allocated_bytes << " = " << total_allocated_bytes << " bytes" << std::endl;

	// Allocate vectors in host memory
	int *v_in = new int[vector_size];
	int *v_out = new int[vector_size];

	// Initialize input vector
	std::cout << "Random values in [0, " << RAND_MAX << "]" << std::endl;
	std::cout << "Initializing input vector..." << std::endl;
	for (int i = 0; i < vector_size; i++) {
		v_in[i] = std::rand();
	}

	// Allocate vector in device memory
	int *v_device;
	checkCudaErrors(hipMalloc(&v_device, vector_allocated_bytes));

	// Copy vector from host memory to device memory
	checkCudaErrors(hipMemcpy(v_device, v_in, vector_allocated_bytes, hipMemcpyHostToDevice));

	hipEvent_t gpu_start_time, gpu_end_time;
	hipEventCreate(&gpu_start_time);
	hipEventCreate(&gpu_end_time);

	// Invoke kernel
	std::cout << "Sorting on GPU..." << std::endl;

	// https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceMergeSort.html

	// Determine temporary device storage requirements
	void *d_temp_storage = nullptr;
	size_t temp_storage_bytes = 0;
	hipcub::DeviceMergeSort::SortKeys(
		d_temp_storage, temp_storage_bytes,
		v_device, vector_size, CompareOp());

	// Allocate temporary storage
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));

	// Run sorting operation
	checkCudaErrors(hipEventRecord(gpu_start_time));
	hipcub::DeviceMergeSort::SortKeys(
		d_temp_storage, temp_storage_bytes,
		v_device, vector_size, CompareOp());
	checkCudaErrors(hipEventRecord(gpu_end_time));
	checkCudaErrors(hipEventSynchronize(gpu_end_time));

	float gpu_elapsed_time_ms = 0;
	checkCudaErrors(hipEventElapsedTime(&gpu_elapsed_time_ms, gpu_start_time, gpu_end_time));
	std::cout << "Elapsed time: " << gpu_elapsed_time_ms << " ms" << std::endl;

	// Copy result from device memory to host memory
	checkCudaErrors(hipMemcpy(v_out, v_device, vector_allocated_bytes, hipMemcpyDeviceToHost));

	// Free device memory
	checkCudaErrors(hipFree(v_device));
	checkCudaErrors(hipFree(d_temp_storage));

	std::cout << "Checking if array is sorted correctly..." << std::endl;
	if (std::is_sorted(v_out, v_out + vector_size)) {
		std::cout << "Array sorted successfully!" << std::endl;
	} else {
		std::cout << "ERROR: array not sorted correctly!" << std::endl;
	}

	// Free host memory
	delete[] v_in;
	delete[] v_out;
}
